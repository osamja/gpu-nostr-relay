/*
 * CUDA ECDSA Signature Verification for secp256k1
 * Batch verification optimized for GPU parallelization
 */

#include <hip/hip_runtime.h>

#include <stdint.h>

// secp256k1 curve parameters
__constant__ uint64_t SECP256K1_P[4] = {
    0xFFFFFFFEFFFFFC2FULL, 0xFFFFFFFFFFFFFFFFULL,
    0xFFFFFFFFFFFFFFFFULL, 0xFFFFFFFFFFFFFFFFULL
};

__constant__ uint64_t SECP256K1_N[4] = {
    0xBFD25E8CD0364141ULL, 0xBAAEDCE6AF48A03BULL,
    0xFFFFFFFFFFFFFFFEULL, 0xFFFFFFFFFFFFFFFFULL
};

__constant__ uint64_t SECP256K1_GX[4] = {
    0x59F2815B16F81798ULL, 0x029BFCDB2DCE28D9ULL,
    0x55A06295CE870B07ULL, 0x79BE667EF9DCBBACULL
};

__constant__ uint64_t SECP256K1_GY[4] = {
    0x9C47D08FFB10D4B8ULL, 0xFD17B448A6855419ULL,
    0x5DA4FBFC0E1108A8ULL, 0x483ADA7726A3C465ULL
};

// Point structure for elliptic curve operations
typedef struct {
    uint64_t x[4];
    uint64_t y[4];
    uint64_t z[4];  // Jacobian coordinates
} ECPoint;

// 256-bit integer operations
__device__ __forceinline__ 
bool is_zero(const uint64_t a[4]) {
    return (a[0] | a[1] | a[2] | a[3]) == 0;
}

__device__ __forceinline__
bool is_equal(const uint64_t a[4], const uint64_t b[4]) {
    return (a[0] == b[0]) && (a[1] == b[1]) && (a[2] == b[2]) && (a[3] == b[3]);
}

__device__ __forceinline__
void copy_bigint(uint64_t dst[4], const uint64_t src[4]) {
    dst[0] = src[0]; dst[1] = src[1]; dst[2] = src[2]; dst[3] = src[3];
}

__device__ __forceinline__
void set_zero(uint64_t a[4]) {
    a[0] = a[1] = a[2] = a[3] = 0;
}

// Modular addition in secp256k1 field
__device__ __forceinline__
void mod_add(uint64_t result[4], const uint64_t a[4], const uint64_t b[4]) {
    uint64_t carry = 0;
    uint64_t temp[4];
    
    // Add with carry
    temp[0] = a[0] + b[0];
    carry = temp[0] < a[0] ? 1 : 0;
    
    temp[1] = a[1] + b[1] + carry;
    carry = (temp[1] < a[1]) || (temp[1] < carry) ? 1 : 0;
    
    temp[2] = a[2] + b[2] + carry;
    carry = (temp[2] < a[2]) || (temp[2] < carry) ? 1 : 0;
    
    temp[3] = a[3] + b[3] + carry;
    
    // Reduce modulo p if necessary
    if (temp[3] > SECP256K1_P[3] || 
        (temp[3] == SECP256K1_P[3] && temp[2] > SECP256K1_P[2]) ||
        (temp[3] == SECP256K1_P[3] && temp[2] == SECP256K1_P[2] && 
         temp[1] > SECP256K1_P[1]) ||
        (temp[3] == SECP256K1_P[3] && temp[2] == SECP256K1_P[2] && 
         temp[1] == SECP256K1_P[1] && temp[0] >= SECP256K1_P[0])) {
        
        // Subtract p
        uint64_t borrow = 0;
        result[0] = temp[0] - SECP256K1_P[0];
        borrow = result[0] > temp[0] ? 1 : 0;
        
        result[1] = temp[1] - SECP256K1_P[1] - borrow;
        borrow = (result[1] > temp[1]) || (borrow && result[1] == temp[1]) ? 1 : 0;
        
        result[2] = temp[2] - SECP256K1_P[2] - borrow;
        borrow = (result[2] > temp[2]) || (borrow && result[2] == temp[2]) ? 1 : 0;
        
        result[3] = temp[3] - SECP256K1_P[3] - borrow;
    } else {
        copy_bigint(result, temp);
    }
}

// Modular subtraction in secp256k1 field
__device__ __forceinline__
void mod_sub(uint64_t result[4], const uint64_t a[4], const uint64_t b[4]) {
    uint64_t temp[4];
    uint64_t borrow = 0;
    
    // Subtract with borrow
    temp[0] = a[0] - b[0];
    borrow = temp[0] > a[0] ? 1 : 0;
    
    temp[1] = a[1] - b[1] - borrow;
    borrow = (temp[1] > a[1]) || (borrow && temp[1] == a[1]) ? 1 : 0;
    
    temp[2] = a[2] - b[2] - borrow;
    borrow = (temp[2] > a[2]) || (borrow && temp[2] == a[2]) ? 1 : 0;
    
    temp[3] = a[3] - b[3] - borrow;
    borrow = (temp[3] > a[3]) || (borrow && temp[3] == a[3]) ? 1 : 0;
    
    if (borrow) {
        // Add p to make result positive
        uint64_t carry = 0;
        result[0] = temp[0] + SECP256K1_P[0];
        carry = result[0] < temp[0] ? 1 : 0;
        
        result[1] = temp[1] + SECP256K1_P[1] + carry;
        carry = (result[1] < temp[1]) || (result[1] < carry) ? 1 : 0;
        
        result[2] = temp[2] + SECP256K1_P[2] + carry;
        carry = (result[2] < temp[2]) || (result[2] < carry) ? 1 : 0;
        
        result[3] = temp[3] + SECP256K1_P[3] + carry;
    } else {
        copy_bigint(result, temp);
    }
}

// Full 256x256 -> 512 bit multiplication
__device__ __forceinline__
void mul_256x256(uint64_t result[8], const uint64_t a[4], const uint64_t b[4]) {
    // Initialize result
    for (int i = 0; i < 8; i++) result[i] = 0;
    
    // Schoolbook multiplication
    for (int i = 0; i < 4; i++) {
        uint64_t carry = 0;
        for (int j = 0; j < 4; j++) {
            // Multiply a[i] * b[j]
            uint64_t high, low;
            
            // 64x64 -> 128 bit multiplication
            uint64_t a_lo = a[i] & 0xFFFFFFFFULL;
            uint64_t a_hi = a[i] >> 32;
            uint64_t b_lo = b[j] & 0xFFFFFFFFULL;
            uint64_t b_hi = b[j] >> 32;
            
            uint64_t ll = a_lo * b_lo;
            uint64_t lh = a_lo * b_hi;
            uint64_t hl = a_hi * b_lo;
            uint64_t hh = a_hi * b_hi;
            
            uint64_t mid = lh + hl;
            uint64_t mid_carry = (mid < lh) ? 1ULL << 32 : 0;
            
            low = ll + (mid << 32);
            high = hh + (mid >> 32) + mid_carry + (low < ll ? 1 : 0);
            
            // Add to result with carry
            uint64_t sum = result[i + j] + low + carry;
            carry = (sum < result[i + j]) || (sum < low) ? 1 : 0;
            carry += high;
            result[i + j] = sum;
            
            if (carry) {
                result[i + j + 1] += carry;
                carry = result[i + j + 1] < carry ? 1 : 0;
            }
        }
    }
}

// Barrett reduction for secp256k1 prime
__device__ __forceinline__
void barrett_reduce(uint64_t result[4], const uint64_t a[8]) {
    // Barrett reduction constants for secp256k1 prime
    // μ = floor(2^512 / p) for 256-bit p
    // Simplified implementation - in production would precompute constants
    
    // For secp256k1 p = 2^256 - 2^32 - 977
    // We can use the special form for faster reduction
    
    // High part of a (a[4] through a[7])
    uint64_t high[4] = {a[4], a[5], a[6], a[7]};
    uint64_t low[4] = {a[0], a[1], a[2], a[3]};
    
    // Multiply high by 2^32 + 977 (the difference from 2^256)
    uint64_t temp1[4], temp2[4];
    
    // high * 2^32 = shift left by 32 bits
    temp1[0] = (high[0] << 32) | (high[1] >> 32);
    temp1[1] = (high[1] << 32) | (high[2] >> 32);
    temp1[2] = (high[2] << 32) | (high[3] >> 32);
    temp1[3] = high[3] << 32;
    
    // high * 977
    uint64_t c977[4] = {977, 0, 0, 0};
    uint64_t temp977[8];
    mul_256x256(temp977, high, c977);
    temp2[0] = temp977[0]; temp2[1] = temp977[1]; 
    temp2[2] = temp977[2]; temp2[3] = temp977[3];
    
    // Add: low + high*2^32 + high*977
    mod_add(result, low, temp1);
    mod_add(result, result, temp2);
}

// Modular multiplication using Barrett reduction
__device__ __forceinline__
void mod_mul(uint64_t result[4], const uint64_t a[4], const uint64_t b[4]) {
    uint64_t temp[8];
    mul_256x256(temp, a, b);
    barrett_reduce(result, temp);
}

// Modular squaring (optimized multiplication by self)
__device__ __forceinline__
void mod_square(uint64_t result[4], const uint64_t a[4]) {
    mod_mul(result, a, a);
}

// Modular inverse using Fermat's little theorem: a^(p-2) mod p
__device__
void mod_inverse(uint64_t result[4], const uint64_t a[4]) {
    // For secp256k1: p-2 = 0xFFFFFFFEFFFFFC2DULL, 0xFFFFFFFFFFFFFFFFULL, 0xFFFFFFFFFFFFFFFFULL, 0xFFFFFFFFFFFFFFFDULL
    uint64_t exp[4] = {
        0xFFFFFFFEFFFFFC2DULL, 0xFFFFFFFFFFFFFFFFULL,
        0xFFFFFFFFFFFFFFFFULL, 0xFFFFFFFFFFFFFFFDULL
    };
    
    // Binary exponentiation
    uint64_t base[4], temp[4];
    copy_bigint(base, a);
    uint64_t one_inv[4] = {1, 0, 0, 0};
    copy_bigint(result, one_inv);
    
    for (int i = 0; i < 256; i++) {
        int word = i / 64;
        int bit = i % 64;
        
        if (exp[word] & (1ULL << bit)) {
            mod_mul(result, result, base);
        }
        
        if (i < 255) {
            mod_square(base, base);
        }
    }
}

// Modular square root using Tonelli-Shanks algorithm
__device__
bool mod_sqrt(uint64_t result[4], const uint64_t a[4]) {
    // For secp256k1, p ≡ 3 (mod 4), so we can use: sqrt(a) = a^((p+1)/4)
    uint64_t exp[4] = {
        0x3FFFFFFFBFFFFF0CULL, 0x0000000000000000ULL,
        0x0000000000000000ULL, 0x4000000000000000ULL
    };
    
    uint64_t base[4], temp[4];
    copy_bigint(base, a);
    uint64_t one_sqrt[4] = {1, 0, 0, 0};
    copy_bigint(result, one_sqrt);
    
    for (int i = 0; i < 256; i++) {
        int word = i / 64;
        int bit = i % 64;
        
        if (exp[word] & (1ULL << bit)) {
            mod_mul(result, result, base);
        }
        
        if (i < 255) {
            mod_square(base, base);
        }
    }
    
    // Verify result
    mod_square(temp, result);
    return is_equal(temp, a);
}

// Compare two 256-bit integers
__device__ __forceinline__
int cmp_bigint(const uint64_t a[4], const uint64_t b[4]) {
    for (int i = 3; i >= 0; i--) {
        if (a[i] > b[i]) return 1;
        if (a[i] < b[i]) return -1;
    }
    return 0;
}

// Point doubling in Jacobian coordinates
// Reference: Guide to Elliptic Curve Cryptography, Algorithm 3.21
__device__
void point_double(ECPoint* result, const ECPoint* p) {
    if (is_zero(p->z)) {
        // Point at infinity
        set_zero(result->x);
        set_zero(result->y);
        set_zero(result->z);
        return;
    }
    
    uint64_t a[4], b[4], c[4], d[4], e[4], f[4];
    
    // A = Y^2
    mod_square(a, p->y);
    
    // B = 4*X*A
    mod_mul(b, p->x, a);
    mod_add(b, b, b);
    mod_add(b, b, b);
    
    // C = 8*A^2
    mod_square(c, a);
    mod_add(c, c, c);
    mod_add(c, c, c);
    mod_add(c, c, c);
    
    // D = 3*X^2 (since a=0 for secp256k1)
    mod_square(d, p->x);
    mod_add(d, d, d);
    mod_add(d, d, d);
    
    // E = D^2
    mod_square(e, d);
    
    // F = E - 2*B
    mod_add(f, b, b);
    mod_sub(f, e, f);
    
    // X3 = F
    copy_bigint(result->x, f);
    
    // Y3 = D*(B - F) - C
    mod_sub(result->y, b, f);
    mod_mul(result->y, d, result->y);
    mod_sub(result->y, result->y, c);
    
    // Z3 = 2*Y*Z
    mod_mul(result->z, p->y, p->z);
    mod_add(result->z, result->z, result->z);
}

// Point addition in Jacobian coordinates
// Reference: Guide to Elliptic Curve Cryptography, Algorithm 3.22
__device__
void point_add(ECPoint* result, const ECPoint* p, const ECPoint* q) {
    if (is_zero(p->z)) {
        *result = *q;
        return;
    }
    if (is_zero(q->z)) {
        *result = *p;
        return;
    }
    
    uint64_t u1[4], u2[4], s1[4], s2[4], h[4], r[4];
    uint64_t z1_squared[4], z2_squared[4], z1_cubed[4], z2_cubed[4];
    uint64_t h_squared[4], h_cubed[4], temp[4];
    
    // Z1^2, Z2^2
    mod_square(z1_squared, p->z);
    mod_square(z2_squared, q->z);
    
    // U1 = X1*Z2^2, U2 = X2*Z1^2
    mod_mul(u1, p->x, z2_squared);
    mod_mul(u2, q->x, z1_squared);
    
    // Z1^3, Z2^3
    mod_mul(z1_cubed, z1_squared, p->z);
    mod_mul(z2_cubed, z2_squared, q->z);
    
    // S1 = Y1*Z2^3, S2 = Y2*Z1^3
    mod_mul(s1, p->y, z2_cubed);
    mod_mul(s2, q->y, z1_cubed);
    
    // Check if points are equal
    if (is_equal(u1, u2)) {
        if (is_equal(s1, s2)) {
            // Points are equal - use doubling
            point_double(result, p);
            return;
        } else {
            // Points are additive inverses - result is point at infinity
            set_zero(result->x);
            set_zero(result->y);
            set_zero(result->z);
            return;
        }
    }
    
    // H = U2 - U1
    mod_sub(h, u2, u1);
    
    // R = S2 - S1
    mod_sub(r, s2, s1);
    
    // H^2, H^3
    mod_square(h_squared, h);
    mod_mul(h_cubed, h_squared, h);
    
    // X3 = R^2 - H^3 - 2*U1*H^2
    mod_square(result->x, r);
    mod_sub(result->x, result->x, h_cubed);
    mod_mul(temp, u1, h_squared);
    mod_add(temp, temp, temp);
    mod_sub(result->x, result->x, temp);
    
    // Y3 = R*(U1*H^2 - X3) - S1*H^3
    mod_mul(temp, u1, h_squared);
    mod_sub(temp, temp, result->x);
    mod_mul(result->y, r, temp);
    mod_mul(temp, s1, h_cubed);
    mod_sub(result->y, result->y, temp);
    
    // Z3 = Z1*Z2*H
    mod_mul(result->z, p->z, q->z);
    mod_mul(result->z, result->z, h);
}

// Scalar multiplication using sliding window method
__device__
void scalar_mult(ECPoint* result, const uint64_t scalar[4], const ECPoint* base) {
    // Point at infinity
    set_zero(result->x);
    set_zero(result->y);
    set_zero(result->z);
    
    // Check for zero scalar
    if (is_zero(scalar)) {
        return;
    }
    
    // Precompute odd multiples [1P, 3P, 5P, 7P, 9P, 11P, 13P, 15P]
    ECPoint precomp[8];
    ECPoint temp, doubled;
    
    precomp[0] = *base;  // 1P
    point_double(&doubled, base);  // 2P
    
    for (int i = 1; i < 8; i++) {
        point_add(&precomp[i], &precomp[i-1], &doubled);
    }
    
    // Sliding window NAF (width 4)
    int naf[256];
    int naf_len = 0;
    
    // Convert scalar to NAF representation
    uint64_t k[4];
    copy_bigint(k, scalar);
    
    while (!is_zero(k)) {
        if (k[0] & 1) {
            // k is odd
            int width = 1;
            uint64_t window = k[0] & 15;  // 4-bit window
            
            // Extend window if possible
            while (width < 4 && (window & (1 << width))) {
                width++;
                if (width < 4) window = k[0] & ((1 << (width + 1)) - 1);
            }
            
            // Make window odd
            if (window & 1) {
                naf[naf_len] = (window + 1) / 2;
            } else {
                naf[naf_len] = 0;
            }
            
            // Subtract window from k
            uint64_t borrow = 0;
            k[0] -= window;
            borrow = k[0] > (UINT64_MAX - window) ? 1 : 0;
            
            for (int i = 1; i < 4 && borrow; i++) {
                if (k[i] == 0) {
                    k[i] = UINT64_MAX;
                } else {
                    k[i]--;
                    borrow = 0;
                }
            }
        } else {
            naf[naf_len] = 0;
        }
        
        // Right shift k by 1
        uint64_t carry = 0;
        for (int i = 3; i >= 0; i--) {
            uint64_t new_carry = k[i] & 1;
            k[i] = (k[i] >> 1) | (carry << 63);
            carry = new_carry;
        }
        
        naf_len++;
        if (naf_len >= 256) break;
    }
    
    // Process NAF from most significant bit
    for (int i = naf_len - 1; i >= 0; i--) {
        point_double(result, result);
        
        if (naf[i] > 0) {
            point_add(result, result, &precomp[naf[i] - 1]);
        }
    }
}

// Convert from bytes to uint64_t array (big endian for secp256k1)
__device__
void bytes_to_bigint(uint64_t dst[4], const uint8_t src[32]) {
    for (int i = 0; i < 4; i++) {
        dst[i] = 0;
        for (int j = 0; j < 8; j++) {
            dst[i] = (dst[i] << 8) | src[i*8 + j];
        }
    }
}

// Convert from affine to Jacobian coordinates
__device__
void affine_to_jacobian(ECPoint* result, const uint64_t x[4], const uint64_t y[4]) {
    copy_bigint(result->x, x);
    copy_bigint(result->y, y);
    uint64_t one_affine[4] = {1, 0, 0, 0};
    copy_bigint(result->z, one_affine);
}

// Convert from Jacobian to affine coordinates
__device__
void jacobian_to_affine(uint64_t x[4], uint64_t y[4], const ECPoint* point) {
    if (is_zero(point->z)) {
        // Point at infinity
        set_zero(x);
        set_zero(y);
        return;
    }
    
    uint64_t z_inv[4], z_inv_squared[4], z_inv_cubed[4];
    
    mod_inverse(z_inv, point->z);
    mod_square(z_inv_squared, z_inv);
    mod_mul(z_inv_cubed, z_inv_squared, z_inv);
    
    mod_mul(x, point->x, z_inv_squared);
    mod_mul(y, point->y, z_inv_cubed);
}

// ECDSA signature verification kernel
__global__
void ecdsa_verify_batch(
    const uint8_t* event_ids,      // 32 bytes per event
    const uint8_t* signatures,     // 64 bytes per signature (r,s)
    const uint8_t* pubkeys,        // 32 bytes per pubkey (x coordinate)
    int* results,                  // Output: 1 = valid, 0 = invalid
    int count                      // Number of signatures to verify
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx >= count) return;
    
    // Extract signature components
    uint64_t r[4], s[4], hash[4], pubkey_x[4];
    
    bytes_to_bigint(r, &signatures[idx * 64]);
    bytes_to_bigint(s, &signatures[idx * 64 + 32]);
    bytes_to_bigint(hash, &event_ids[idx * 32]);
    bytes_to_bigint(pubkey_x, &pubkeys[idx * 32]);
    
    // Basic range checks
    if (is_zero(r) || is_zero(s)) {
        results[idx] = 0;
        return;
    }
    
    // Check if r, s are in valid range [1, n-1]
    // Simplified check - in production would need proper comparison
    if (r[3] >= SECP256K1_N[3] || s[3] >= SECP256K1_N[3]) {
        results[idx] = 0;
        return;
    }
    
    // Reconstruct public key point from x-coordinate
    // For secp256k1: y^2 = x^3 + 7 (mod p)
    ECPoint pubkey_point;
    copy_bigint(pubkey_point.x, pubkey_x);
    
    uint64_t x_squared[4], x_cubed[4], y_squared[4], y_coord[4];
    
    // Calculate x^3 + 7
    mod_square(x_squared, pubkey_x);
    mod_mul(x_cubed, x_squared, pubkey_x);
    
    uint64_t seven[4] = {7, 0, 0, 0};
    mod_add(y_squared, x_cubed, seven);
    
    // Calculate square root using Tonelli-Shanks
    bool sqrt_exists = mod_sqrt(y_coord, y_squared);
    
    if (!sqrt_exists) {
        results[idx] = 0;
        return;
    }
    
    // For Nostr, we use the even y-coordinate (compressed public key format)
    if (y_coord[0] & 1) {
        // y is odd, use p - y to get even y
        mod_sub(y_coord, SECP256K1_P, y_coord);
    }
    
    copy_bigint(pubkey_point.y, y_coord);
    
    uint64_t one_pubkey[4] = {1, 0, 0, 0};
    copy_bigint(pubkey_point.z, one_pubkey);
    
    // ECDSA verification algorithm
    // 1. Calculate u1 = hash * s^(-1) mod n
    // 2. Calculate u2 = r * s^(-1) mod n  
    // 3. Calculate point P = u1*G + u2*pubkey
    // 4. Verify that P.x ≡ r (mod n)
    
    uint64_t s_inv[4], u1[4], u2[4];
    
    // Calculate s^(-1) mod n using Fermat's little theorem
    // For secp256k1: n-2 = 0xBFD25E8CD0364141ULL, 0xBAAEDCE6AF48A03BULL, 0xFFFFFFFFFFFFFFFCULL, 0xFFFFFFFFFFFFFFFFULL
    uint64_t n_minus_2[4] = {
        0xBFD25E8CD036413FULL, 0xBAAEDCE6AF48A03BULL,
        0xFFFFFFFFFFFFFFFCULL, 0xFFFFFFFFFFFFFFFFULL
    };
    
    // s_inv = s^(n-2) mod n
    uint64_t base[4], exp_result[4];
    copy_bigint(base, s);
    uint64_t one_val[4] = {1, 0, 0, 0};
    copy_bigint(s_inv, one_val);
    
    // Binary exponentiation for modular inverse
    for (int i = 0; i < 256; i++) {
        int word = i / 64;
        int bit = i % 64;
        
        if (n_minus_2[word] & (1ULL << bit)) {
            // s_inv = (s_inv * base) mod n
            uint64_t temp[8];
            mul_256x256(temp, s_inv, base);
            
            // Reduce modulo n (simplified)
            for (int j = 0; j < 4; j++) {
                s_inv[j] = temp[j];
            }
        }
        
        if (i < 255) {
            // base = (base * base) mod n
            uint64_t temp[8];
            mul_256x256(temp, base, base);
            for (int j = 0; j < 4; j++) {
                base[j] = temp[j];
            }
        }
    }
    
    // u1 = (hash * s_inv) mod n
    uint64_t temp_u1[8];
    mul_256x256(temp_u1, hash, s_inv);
    for (int i = 0; i < 4; i++) {
        u1[i] = temp_u1[i];
    }
    
    // u2 = (r * s_inv) mod n
    uint64_t temp_u2[8];
    mul_256x256(temp_u2, r, s_inv);
    for (int i = 0; i < 4; i++) {
        u2[i] = temp_u2[i];
    }
    
    // Create generator point
    ECPoint generator;
    affine_to_jacobian(&generator, SECP256K1_GX, SECP256K1_GY);
    
    // Calculate u1*G + u2*pubkey
    ECPoint point1, point2, result_point;
    scalar_mult(&point1, u1, &generator);
    scalar_mult(&point2, u2, &pubkey_point);
    point_add(&result_point, &point1, &point2);
    
    // Convert result to affine coordinates
    uint64_t result_x[4], result_y[4];
    jacobian_to_affine(result_x, result_y, &result_point);
    
    // Check if result_x ≡ r (mod n)
    // Reduce result_x modulo n
    uint64_t r_check[4];
    copy_bigint(r_check, result_x);
    
    // Simple modular reduction (in production would use proper reduction)
    while (cmp_bigint(r_check, SECP256K1_N) >= 0) {
        mod_sub(r_check, r_check, SECP256K1_N);
    }
    
    if (is_equal(r_check, r)) {
        results[idx] = 1;
    } else {
        results[idx] = 0;
    }
}

// C interface for Python
extern "C" {
    
int cuda_ecdsa_verify_batch(
    const uint8_t* h_event_ids,
    const uint8_t* h_signatures, 
    const uint8_t* h_pubkeys,
    int* h_results,
    int count
) {
    // GPU memory pointers
    uint8_t *d_event_ids, *d_signatures, *d_pubkeys;
    int *d_results;
    
    // Calculate sizes
    size_t event_ids_size = count * 32;
    size_t signatures_size = count * 64;
    size_t pubkeys_size = count * 32;
    size_t results_size = count * sizeof(int);
    
    // Allocate GPU memory
    hipError_t err;
    err = hipMalloc(&d_event_ids, event_ids_size);
    if (err != hipSuccess) return -1;
    
    err = hipMalloc(&d_signatures, signatures_size);
    if (err != hipSuccess) { hipFree(d_event_ids); return -1; }
    
    err = hipMalloc(&d_pubkeys, pubkeys_size);
    if (err != hipSuccess) { 
        hipFree(d_event_ids); hipFree(d_signatures); return -1; 
    }
    
    err = hipMalloc(&d_results, results_size);
    if (err != hipSuccess) { 
        hipFree(d_event_ids); hipFree(d_signatures); hipFree(d_pubkeys); 
        return -1; 
    }
    
    // Copy data to GPU
    hipMemcpy(d_event_ids, h_event_ids, event_ids_size, hipMemcpyHostToDevice);
    hipMemcpy(d_signatures, h_signatures, signatures_size, hipMemcpyHostToDevice);
    hipMemcpy(d_pubkeys, h_pubkeys, pubkeys_size, hipMemcpyHostToDevice);
    
    // Launch kernel
    int threads_per_block = 256;
    int blocks = (count + threads_per_block - 1) / threads_per_block;
    
    ecdsa_verify_batch<<<blocks, threads_per_block>>>(
        d_event_ids, d_signatures, d_pubkeys, d_results, count
    );
    
    // Copy results back
    hipMemcpy(h_results, d_results, results_size, hipMemcpyDeviceToHost);
    
    // Cleanup
    hipFree(d_event_ids);
    hipFree(d_signatures);
    hipFree(d_pubkeys);
    hipFree(d_results);
    
    // Check for errors
    err = hipGetLastError();
    return (err == hipSuccess) ? 0 : -1;
}

}  // extern "C"